#include "VolumeManagerKernel.h"
#include "defs.h"
#include "ritkCudaMacros.h"
#include <stdio.h>


//----------------------------------------------------------------------------
extern "C"
void CUDAAddPointsToVolumePointToVoxel(float* points, unsigned char* voxels, float* config, unsigned long numPts)
{
	kernelAddPointsToVolumePointToVoxel<<<DIVUP(numPts, CUDA_THREADS_PER_BLOCK), CUDA_THREADS_PER_BLOCK>>>(points, voxels, config, numPts);
}