/* This file is part of the Random Ball Cover (RBC) library.
 * (C) Copyright 2010, Lawrence Cayton [lcayton@tuebingen.mpg.de]
 */

#ifndef BRUTE_CU
#define BRUTE_CU

#include "utilsGPU.h"
#include "utils.h"
#include "rbc.h"
#include "defsRBC.h"
#include "kernels.h"
#include "kernelWrap.h"
#include "brute.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
//#include<gsl/gsl_sort.h>

void bruteRangeCount(matrix x, matrix q, real *ranges, unint *cnts){
  matrix dx, dq;
  real *dranges;
  unint *dcnts;
  
  copyAndMove(&dx, &x);
  copyAndMove(&dq, &q);

  ( hipMalloc( (void**)&dranges, q.pr*sizeof(*dranges) ) );
  hipMemcpy( dranges, ranges, q.r*sizeof(*dranges), hipMemcpyHostToDevice );

  ( hipMalloc( (void**)&dcnts, q.pr*sizeof(*dcnts) ) );
  
  rangeCountWrap(dq, dx, dranges, dcnts);
  
  hipMemcpy(cnts, dcnts, q.r*sizeof(*cnts), hipMemcpyDeviceToHost );

  hipFree(dcnts);
  hipFree(dranges);
  hipFree(dx.mat);
  hipFree(dq.mat);
}


void bruteSearch(matrix x, matrix q, unint *NNs){
  real *dMins;
  unint *dMinIDs;
  matrix dx, dq;

  
  dx.r=x.r; dx.pr=x.pr; dx.c=x.c; dx.pc=x.pc; dx.ld=x.ld;
  dq.r=q.r; dq.pr=q.pr; dq.c=q.c; dq.pc=q.pc; dq.ld=q.ld;

  ( hipMalloc((void**)&dMins, q.pr*sizeof(*dMins)) );
  ( hipMalloc((void**)&dMinIDs, q.pr*sizeof(*dMinIDs)) );
  ( hipMalloc((void**)&dx.mat, dx.pr*dx.pc*sizeof(*dx.mat)) );
  ( hipMalloc((void**)&dq.mat, dq.pr*dq.pc*sizeof(*dq.mat)) );

  hipMemcpy(dx.mat,x.mat,x.pr*x.pc*sizeof(*dx.mat),hipMemcpyHostToDevice);
  hipMemcpy(dq.mat,q.mat,q.pr*q.pc*sizeof(*dq.mat),hipMemcpyHostToDevice);
  
  nnWrap(dq,dx,dMins,dMinIDs);

  hipMemcpy(NNs,dMinIDs,dq.r*sizeof(*NNs),hipMemcpyDeviceToHost);
  
  hipFree(dMins);
  hipFree(dMinIDs);
  hipFree(dx.mat);
  hipFree(dq.mat);
}


void bruteK(matrix x, matrix q, intMatrix NNs, matrix NNdists){
  matrix dNNdists;
  intMatrix dMinIDs;
  matrix dx, dq;
  
  dx.r=x.r; dx.pr=x.pr; dx.c=x.c; dx.pc=x.pc; dx.ld=x.ld;
  dq.r=q.r; dq.pr=q.pr; dq.c=q.c; dq.pc=q.pc; dq.ld=q.ld;
  dNNdists.r=q.r; dNNdists.pr=q.pr; dNNdists.c=K; dNNdists.pc=K; dNNdists.ld=dNNdists.pc;
  dMinIDs.r=q.r; dMinIDs.pr=q.pr; dMinIDs.c=K; dMinIDs.pc=K; dMinIDs.ld=dMinIDs.pc;

  ( hipMalloc((void**)&dNNdists.mat, dNNdists.pc*dNNdists.pr*sizeof(*dNNdists.mat)) );
  ( hipMalloc((void**)&dMinIDs.mat, dMinIDs.pc*dMinIDs.pr*sizeof(*dMinIDs.mat)) );
  ( hipMalloc((void**)&dx.mat, dx.pr*dx.pc*sizeof(*dx.mat)) );
  ( hipMalloc((void**)&dq.mat, dq.pr*dq.pc*sizeof(*dq.mat)) );

  hipMemcpy(dx.mat,x.mat,x.pr*x.pc*sizeof(*dx.mat),hipMemcpyHostToDevice);
  hipMemcpy(dq.mat,q.mat,q.pr*q.pc*sizeof(*dq.mat),hipMemcpyHostToDevice);
  
  knnWrap(dq,dx,dNNdists,dMinIDs);

  hipMemcpy(NNs.mat,dMinIDs.mat,NNs.pr*NNs.pc*sizeof(*NNs.mat),hipMemcpyDeviceToHost);
  hipMemcpy(NNdists.mat,dNNdists.mat,NNdists.pr*NNdists.pc*sizeof(*NNdists.mat),hipMemcpyDeviceToHost);

  hipFree(dNNdists.mat);
  hipFree(dMinIDs.mat);
  hipFree(dx.mat);
  hipFree(dq.mat);
}


void bruteCPU(matrix X, matrix Q, unint *NNs){
  real *dtoNNs; 
  real temp;

  unint i, j;

  dtoNNs = (real*)calloc(Q.r,sizeof(*dtoNNs));
  
  for( i=0; i<Q.r; i++ ){
    dtoNNs[i] = MAX_REAL;
    NNs[i] = 0;
    for(j=0; j<X.r; j++ ){
      temp = distVec( Q, X, i, j );
      if( temp < dtoNNs[i]){
	NNs[i] = j;
	dtoNNs[i] = temp;
      }
    }
  }
  
  free(dtoNNs);  
}


//The following method works properly, but requires the GNU scientific
//library.  If you want to use it, uncomment the code, uncomment the include
//above, and adjust the makefile.  It was used for debugging purposes, but
//is not required by anything else.

/* void bruteKCPU(matrix x, matrix q, intMatrix NNs){ */
/*   int i, j; */

/*   float **d; */
/*   d = (float**)calloc(q.pr, sizeof(*d)); */
/*   size_t **t; */
/*   t = (size_t**)calloc(q.pr, sizeof(*t)); */
/*   for( i=0; i<q.pr; i++){ */
/*     d[i] = (float*)calloc(x.pr, sizeof(**d)); */
/*     t[i] = (size_t*)calloc(x.pr, sizeof(**t)); */
/*   } */

/*   for( i=0; i<q.r; i++){ */
/*     for( j=0; j<x.r; j++) */
/*       d[i][j] = distVec( q, x, i, j ); */
/*     gsl_sort_float_index(t[i], d[i], 1, x.r); */
/*     for ( j=0; j<K; j++) */
/*       NNs.mat[IDX( i, j, NNs.ld )] = t[i][j]; */
/*   } */

/*   for( i=0; i<q.pr; i++){ */
/*     free(t[i]); */
/*     free(d[i]); */
/*   } */
/*   free(t); */
/*   free(d); */
/* } */
#endif
