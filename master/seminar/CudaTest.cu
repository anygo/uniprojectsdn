#include "CudaTestKernel.h"

#include <iostream>
#include <stdio.h>
#include <cutil.h>
#include <stdio.h>
#include <cutil_inline.h>
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
#include <hip/channel_descriptor.h>
#include <hip/hip_runtime_api.h>
#include "CudaContext.h"


extern "C"
void cudaTest() {

	printf("Testing Cuda...");
	kernel <<<1,1>>>();
	printf("Tested Cuda...");
	
}