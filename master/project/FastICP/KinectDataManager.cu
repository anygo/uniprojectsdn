#include "KinectDataManagerKernel.h"
#include "defs.h"


//----------------------------------------------------------------------------
extern "C"
void CUDARangeToWorld(float* pointsOut, const hipArray* inputImageArray)
{
	// Set input image texture parameters and bind texture to the array. Texture is defined in the kernel
	InputImageTexture.addressMode[0] = hipAddressModeClamp;
	InputImageTexture.addressMode[1] = hipAddressModeClamp;
	InputImageTexture.filterMode = hipFilterModePoint;
	InputImageTexture.normalized = false;
	hipBindTextureToArray(InputImageTexture, inputImageArray);
	
	// Kernel Invocation
	dim3 DimBlock(16, 16);
	dim3 DimGrid(DivUp(KINECT_IMAGE_WIDTH, DimBlock.x), DivUp(KINECT_IMAGE_HEIGHT, DimBlock.y));
	CUDARangeToWorldKernel<16,16><<<DimGrid,DimBlock>>>(pointsOut);

	// Unbind texture
	hipUnbindTexture(InputImageTexture);
}


//----------------------------------------------------------------------------
extern "C"
void CUDAExtractLandmarks(float* landmarksOut, float* pointsIn, unsigned long* indices, unsigned long numLandmarks)
{
	kernelExtractLandmarks<<<DivUp(numLandmarks, CUDA_THREADS_PER_BLOCK), CUDA_THREADS_PER_BLOCK>>>(landmarksOut, pointsIn, indices, numLandmarks);
}