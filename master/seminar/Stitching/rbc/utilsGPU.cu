/* This file is part of the Random Ball Cover (RBC) library.
 * (C) Copyright 2010, Lawrence Cayton [lcayton@tuebingen.mpg.de]
 */

#ifndef UTILSGPU_CU
#define UTILSGPU_CU

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "defsRBC.h"
#include "utilsGPU.h"

void copyAndMove(matrix *dx, const matrix *x){
  dx->r = x->r; 
  dx->c = x->c;
  dx->pr = x->pr;
  dx->pc = x->pc;
  dx->ld = x->ld;

  ( hipMalloc( (void**)&(dx->mat), dx->pr*dx->pc*sizeof(*(dx->mat)) ) );
  hipMemcpy( dx->mat, x->mat, dx->pr*dx->pc*sizeof(*(dx->mat)), hipMemcpyHostToDevice );
  
}


void copyAndMoveI(intMatrix *dx, const intMatrix *x){
  dx->r = x->r; 
  dx->c = x->c;
  dx->pr = x->pr;
  dx->pc = x->pc;
  dx->ld = x->ld;

  ( hipMalloc( (void**)&(dx->mat), dx->pr*dx->pc*sizeof(*(dx->mat)) ) );
  hipMemcpy( dx->mat, x->mat, dx->pr*dx->pc*sizeof(*(dx->mat)), hipMemcpyHostToDevice );
  
}


void copyAndMoveC(charMatrix *dx, const charMatrix *x){
  dx->r = x->r; 
  dx->c = x->c;
  dx->pr = x->pr;
  dx->pc = x->pc;
  dx->ld = x->ld;

  ( hipMalloc( (void**)&(dx->mat), dx->pr*dx->pc*sizeof(*(dx->mat)) ) );
  hipMemcpy( dx->mat, x->mat, dx->pr*dx->pc*sizeof(*(dx->mat)), hipMemcpyHostToDevice );
  
}



#endif
