#include "hip/hip_runtime.h"
/* This file is part of the Random Ball Cover (RBC) library.
 * (C) Copyright 2010, Lawrence Cayton [lcayton@tuebingen.mpg.de]
 */
#ifndef SKERNELWRAP_CU
#define SKERNELWRAP_CU

#include "sKernel.h"
#include <hip/hip_runtime.h>
#include "defsRBC.h"
#include "utilsGPU.h"
#include <stdio.h>

void getCountsWrap(unint *counts, charMatrix ir, intMatrix sums){
  dim3 block(BLOCK_SIZE,1);
  dim3 grid;
  grid.y=1;
  unint todo, numDone;
  
  numDone = 0;
  while(numDone < ir.pr){
    todo = MIN( ir.pr - numDone, MAX_BS*BLOCK_SIZE );
    grid.x = todo/BLOCK_SIZE;
    getCountsKernel<<<grid,block>>>(counts, numDone, ir, sums);
    numDone += todo;
  }
}


void buildMapWrap(intMatrix map, charMatrix ir, intMatrix sums, unint offSet){
  unint numScans = (ir.c+SCAN_WIDTH-1)/SCAN_WIDTH;
  dim3 block( SCAN_WIDTH/2, 1 );
  dim3 grid;
  unint todo, numDone;

  grid.x = numScans;
  numDone = 0;
  while( numDone < ir.r ){
    todo = MIN( ir.r-numDone, MAX_BS );
    grid.y = todo;
    buildMapKernel<<<grid,block>>>(map, ir, sums, offSet+numDone);
    numDone += todo;
  }
}


void sumWrap(charMatrix in, intMatrix sum){
  int i; 
  unint todo, numDone, temp;
  unint n = in.c;
  unint numScans = (n+SCAN_WIDTH-1)/SCAN_WIDTH;
  unint depth = ceil( log(static_cast<float>(n)) / log(static_cast<float>(SCAN_WIDTH)) ) -1 ;
  unint *width = (unint*)calloc( depth+1, sizeof(*width) );
    
  intMatrix *dAux;
  dAux = (intMatrix*)calloc( depth+1, sizeof(*dAux) );
  
  for( i=0, temp=n; i<=depth; i++){
    temp = (temp+SCAN_WIDTH-1)/SCAN_WIDTH;
    dAux[i].r=dAux[i].pr=in.r; dAux[i].c=dAux[i].pc=dAux[i].ld=temp;
    ( hipMalloc( (void**)&dAux[i].mat, dAux[i].pr*dAux[i].pc*sizeof(*dAux[i].mat) ) );
  }

  dim3 block( SCAN_WIDTH/2, 1 );
  dim3 grid;
  
  numDone=0;
  while( numDone < in.r ){
    todo = MIN( in.r - numDone, MAX_BS );
    numScans = (n+SCAN_WIDTH-1)/SCAN_WIDTH;
    dAux[0].r=dAux[0].pr=todo;
    grid.x = numScans;
    grid.y = todo;
    sumKernel<<<grid,block>>>(in, sum, dAux[0], n);
    hipDeviceSynchronize();
    
    width[0] = numScans; // Necessary because following loop might not be entered
    for( i=0; i<depth; i++ ){
      width[i] = numScans;
      numScans = (numScans+SCAN_WIDTH-1)/SCAN_WIDTH;
      dAux[i+1].r=dAux[i+1].pr=todo;
      
      grid.x = numScans;
      sumKernelI<<<grid,block>>>(dAux[i], dAux[i], dAux[i+1], width[i]);
      hipDeviceSynchronize();
    }
  
    for( i=depth-1; i>0; i-- ){
      grid.x = width[i];
      combineSumKernel<<<grid,block>>>(dAux[i-1], numDone, dAux[i], width[i-1]);
      hipDeviceSynchronize();
    }
    
    grid.x = width[0];
    combineSumKernel<<<grid,block>>>(sum, numDone, dAux[0], n);
    hipDeviceSynchronize();
    
    numDone += todo;
  }

  for( i=0; i<=depth; i++)
   hipFree(dAux[i].mat);
  free(dAux);
  free(width);
}


#endif
