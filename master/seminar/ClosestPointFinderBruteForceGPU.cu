#include "ClosestPointFinderBruteForceGPUKernel.h"
#include "defs.h"

#include <stdio.h>
#include <cutil.h>
#include <stdio.h>
#include <cutil_inline.h>
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
#include <hip/channel_descriptor.h>
#include <hip/hip_runtime_api.h>


// global pointers for gpu... 
int* dev_indices;
Point6D* dev_source;
Point6D* dev_target;
	

extern "C"
void initGPU(Point6D* target, int nrOfPoints) 
{
	// allocate memory on gpu
	hipMalloc((void**)&dev_indices, nrOfPoints*sizeof(int));
	hipMalloc((void**)&dev_source, nrOfPoints*sizeof(Point6D));
	hipMalloc((void**)&dev_target, nrOfPoints*sizeof(Point6D));
	
	hipMemcpy(dev_target, target, nrOfPoints*sizeof(Point6D), hipMemcpyHostToDevice);
}

extern "C"
void cleanupGPU() 
{
	// free memory
	hipFree(dev_indices);
	hipFree(dev_source);
	hipFree(dev_target);
}

extern "C"
void FindClosestPointsCUDA(int nrOfPoints, int metric, bool useRGBData, double weightRGB, int* indices, Point6D* source)
{

	// copy data from host to gpu
	hipMemcpy(dev_source, source, nrOfPoints*sizeof(Point6D), hipMemcpyHostToDevice);	

	// execution
	if (useRGBData)
		kernelWithRGB<<<nrOfPoints,1>>>(nrOfPoints, metric, (float)weightRGB, dev_indices, dev_source, dev_target);
	else
		kernelWithoutRGB<<<nrOfPoints,1>>>(nrOfPoints, metric, dev_indices, dev_source, dev_target);
			
	// copy data from gpu to host
	hipMemcpy(indices, dev_indices, nrOfPoints*sizeof(int), hipMemcpyDeviceToHost);	
	
}