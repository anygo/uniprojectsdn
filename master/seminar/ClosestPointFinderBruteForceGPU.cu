#include "ClosestPointFinderBruteForceGPUKernel.h"
#include "defs.h"

#include <cutil_inline.h>


// we have to copy the source points only once, because they will be
// transformed directly on the gpu! unfortunately, we do not yet have
// the source points, hence we use that boolean to determine whether
// the data is already on the gpu (after 1st iteration)
bool sourceCopied;
	

extern "C"
void initGPU(PointCoords* targetCoords, PointColors* targetColors, int nrOfPoints) 
{
	sourceCopied = false;

	// allocate memory on gpu
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_indices, nrOfPoints*sizeof(unsigned short)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_sourceCoords, nrOfPoints*sizeof(PointCoords)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_targetCoords, nrOfPoints*sizeof(PointCoords)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_sourceColors, nrOfPoints*sizeof(PointColors)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_targetColors, nrOfPoints*sizeof(PointColors)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_distances, nrOfPoints*sizeof(float)));
	
	CUDA_SAFE_CALL(hipMemcpy(dev_targetCoords, targetCoords, nrOfPoints*sizeof(PointCoords), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dev_targetColors, targetColors, nrOfPoints*sizeof(PointColors), hipMemcpyHostToDevice));
}

extern "C"
void cleanupGPU() 
{
	// free memory
	CUDA_SAFE_CALL(hipFree(dev_indices));
	CUDA_SAFE_CALL(hipFree(dev_sourceCoords));
	CUDA_SAFE_CALL(hipFree(dev_targetCoords));
	CUDA_SAFE_CALL(hipFree(dev_sourceColors));
	CUDA_SAFE_CALL(hipFree(dev_targetColors));
	
	CUDA_SAFE_CALL(hipFree(dev_distances));
}

extern "C"
void FindClosestPointsCUDA(int nrOfPoints, int metric, bool useRGBData, float weightRGB, unsigned short* indices, PointCoords* sourceCoords, PointColors* sourceColors, float* distances)
{
	// copy data from host to gpu only if it is not yet copied
	// copy only once, because the data is transformed directly on the gpu!
	if (!sourceCopied)
		CUDA_SAFE_CALL(hipMemcpy(dev_sourceCoords, sourceCoords, nrOfPoints*sizeof(PointCoords), hipMemcpyHostToDevice));	
		CUDA_SAFE_CALL(hipMemcpy(dev_sourceColors, sourceColors, nrOfPoints*sizeof(PointColors), hipMemcpyHostToDevice));	
	sourceCopied = true;

	// find the closest point for each pixel
	if (useRGBData)
		kernelWithRGB<<<nrOfPoints,1>>>(nrOfPoints, metric, weightRGB, dev_indices, dev_sourceCoords, dev_sourceColors, dev_targetCoords, dev_targetColors, dev_distances);
	else
		kernelWithoutRGB<<<nrOfPoints,1>>>(nrOfPoints, metric, dev_indices, dev_sourceCoords, dev_targetCoords, dev_distances);
		
	CUT_CHECK_ERROR("Kernel execution failed (while trying to find closest points)");
			
	// copy data from gpu to host
	CUDA_SAFE_CALL(hipMemcpy(indices, dev_indices, nrOfPoints*sizeof(unsigned short), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(distances, dev_distances, nrOfPoints*sizeof(float), hipMemcpyDeviceToHost));
}

extern "C"
void TransformPointsDirectlyOnGPU(int nrOfPoints, double transformationMatrix[4][4], PointCoords* writeTo, float* distances)
{
	// allocate memory for transformation matrix (will be stored linearly) and copy it
	float tmp[16];
	tmp[0] = (float)transformationMatrix[0][0];
	tmp[1] = (float)transformationMatrix[0][1];
	tmp[2] = (float)transformationMatrix[0][2];
	tmp[3] = (float)transformationMatrix[0][3];
	tmp[4] = (float)transformationMatrix[1][0];
	tmp[5] = (float)transformationMatrix[1][1];
	tmp[6] = (float)transformationMatrix[1][2];
	tmp[7] = (float)transformationMatrix[1][3];
	tmp[8] = (float)transformationMatrix[2][0];
	tmp[9] = (float)transformationMatrix[2][1];
	tmp[10] = (float)transformationMatrix[2][2];
	tmp[11] = (float)transformationMatrix[2][3];
	tmp[12] = (float)transformationMatrix[3][0];
	tmp[13] = (float)transformationMatrix[3][1];
	tmp[14] = (float)transformationMatrix[3][2];
	tmp[15] = (float)transformationMatrix[3][3];
	
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_transformationMatrix), tmp, 16*sizeof(float), 0));
	
	// compute transformations
	kernelTransformPointsAndComputeDistance<<<nrOfPoints,1>>>(dev_sourceCoords, dev_distances);
	CUT_CHECK_ERROR("Kernel execution failed (while transforming points)");
	
	// copy distance array to host
	CUDA_SAFE_CALL(hipMemcpy(distances, dev_distances, nrOfPoints*sizeof(float), hipMemcpyDeviceToHost));
	
	// copy transformed points to host
	CUDA_SAFE_CALL(hipMemcpy(writeTo, dev_sourceCoords, nrOfPoints*sizeof(PointCoords), hipMemcpyDeviceToHost));
}
